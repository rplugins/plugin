    #include <hip/hip_runtime.h>
    #include <iostream>

    using namespace std;

    #define SIZE 7


    __global__ void max(int *a , int *c)
    {
    int i = threadIdx.x;

    *c = a[0];

            if(a[i] > *c)
                    {
                    *c = a[i];
                    }

    }

    int main()
    {
    int i;


    int a[SIZE];
    int c;

    int *dev_a, *dev_c;

    hipMalloc((void **) &dev_a, SIZE*sizeof(int));
    hipMalloc((void **) &dev_c, SIZE*sizeof(int));

    cout<<"Enter the numbers  : \n";
    for( i = 0 ; i < SIZE ; i++)
    {
    	cin>>a[i];
    }
    for( i = 0 ; i < SIZE ; i++)
    {
    	cout<<a[i]<<" ";
    }

    hipMemcpy(dev_a , a, SIZE*sizeof(int),hipMemcpyHostToDevice);
    max<<<1,SIZE>>>(dev_a,dev_c);
    hipMemcpy(&c, dev_c, SIZE*sizeof(int),hipMemcpyDeviceToHost);

    cout<<"\n max value = ";
    cout<<c;

    hipFree(dev_a);
    hipFree(dev_c);


    return 0;
    }





